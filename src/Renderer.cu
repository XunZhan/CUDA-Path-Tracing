#include "hip/hip_runtime.h"
//
// Created by zhanx on 11/10/2020.
//
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <cstdio>
#include <stdlib.h>
#include <assert.h>
#include <cmath>
#include <queue>
#include "Scene.h"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
                  file << ":" << line << " '" << func << "' \n";
        std::cerr << hipGetErrorString(result) << "\n";
        hipDeviceReset(); // Make sure we call CUDA Device Reset before exiting
        exit(99);
    }
}

#define CELL_X_NUM 32 // cell number in x axis
#define CELL_Y_NUM 32 // cell number in y axis


// CPU Memory
static dim3 blocks;
static dim3 threads;
static int num_pixles;
static int num_bvhElems;
static int num_triangles;
static Scene* scene_host = NULL;
BVHElem* bvhElem_host = NULL;
Bounds3* bound_host = NULL;
Triangle* triangle_host = NULL;

// GPU Memory
hiprandState *devStates;
static Ray* ray_dev = NULL;
BVHElem* bvhElem_dev = NULL;
Material* material_dev = NULL;
Triangle* triangle_dev = NULL;
Bounds3* bound_dev = NULL;       // bounding box for objects not triangle

// Unified Memory
float *frameBuffer = NULL;


// cpu internal function definition
void BuildBvhNodeList(Scene* scene);


__device__ Vector3f Material::cudaSample (const Vector3f &wi, const Vector3f &N , hiprandState *state, int pid) {                       // sample a ray by Material properties
    switch(m_type){
        case DIFFUSE:
        {
            // uniform sample on the hemisphere
            hiprandState localState = state[pid];
            hiprand_init((unsigned int) clock64(), pid, 0, &localState);
            float x_1 = hiprand_uniform(&localState);
            hiprand_init((unsigned int) clock64(), pid, 0, &localState);
            float x_2 = hiprand_uniform(&localState);
            float z = fabs(1.0f - 2.0f * x_1);
            float r = sqrt(1.0f - z * z), phi = 2 * M_PI * x_2;
            Vector3f localRay(r*std::cos(phi), r*std::sin(phi), z);
            return toWorld(localRay, N);

            break;
        }
    }
    return Vector3f(0, 0, 0);
}

__device__ void Triangle::CudaSample (Intersection &pos, float &pdf, hiprandState *state, int pid) {
    hiprandState localState = state[pid];
    float x = sqrt(hiprand_uniform(&localState)), y = hiprand_uniform(&localState);
    pos.emit = this->m->m_emission;
    pos.coords = v0 * (1.0f - x) + v1 * (x * (1.0f - y)) + v2 * (x * y);
    pos.normal = this->normal;
    pdf = 1.0f / area;
}



void InitRender(Scene* scene) {

    // thread limit per block is 1024
    blocks = dim3(CELL_X_NUM, CELL_Y_NUM);
    threads = dim3((scene->width + blocks.x - 1) / blocks.x,
                   (scene->height + blocks.y - 1) / blocks.y);

    num_pixles = scene->width * scene->height;
    num_bvhElems = scene->bvh->nodeCount;

    // convert BVH to BVHElem
    BuildBvhNodeList(scene);

    // Load to CPU Memory
    scene_host = scene;


    checkCudaErrors(hipMalloc(&ray_dev, num_pixles*sizeof(Ray)));

    if (num_bvhElems > 0) {
        checkCudaErrors(hipMalloc(&bvhElem_dev, num_bvhElems*sizeof(BVHElem)));
        checkCudaErrors(hipMemcpy(bvhElem_dev,bvhElem_host,
                                   num_bvhElems*sizeof(BVHElem), hipMemcpyHostToDevice));
    }

    if (num_triangles > 0) {
        checkCudaErrors(hipMalloc(&material_dev, num_triangles*sizeof(Material)));
        for (int i = 0; i<num_triangles; i++) {
            checkCudaErrors(hipMemcpy(&material_dev[i], triangle_host[i].m, sizeof(Material), hipMemcpyHostToDevice));
        }
    }

    if (num_triangles > 0) {
        checkCudaErrors(hipMalloc(&triangle_dev, num_triangles*sizeof(Triangle)));
        checkCudaErrors(hipMemcpy(triangle_dev, triangle_host, num_triangles*sizeof(Triangle), hipMemcpyHostToDevice));
        // TODO: allcoate material_dev address to triangle_dev, failed
//        for (int i = 0; i<num_triangles; i++) {
//            checkCudaErrors(hipMemcpy(&triangle_dev[i].m, &material_dev[i], sizeof(Material*), hipMemcpyDeviceToDevice));
//        }
    }

    if (num_bvhElems > 0) {
        checkCudaErrors(hipMalloc(&bound_dev, num_bvhElems*sizeof(Bounds3)));
        checkCudaErrors(hipMemcpy(bound_dev, bound_host,
                                   num_bvhElems*sizeof(Bounds3), hipMemcpyHostToDevice));
    }

    // Init frame buffer in Unified Memory
    checkCudaErrors(hipMallocManaged((void **)&frameBuffer, 3*num_pixles*sizeof(float)));
    for (int i = 0; i<3*num_pixles; i++) {
        frameBuffer[i] = 0;
    }

    // init cuda random generator
    int threadNum = blocks.x * blocks.y * threads.x * threads.y;
    checkCudaErrors(hipMalloc((void **)&devStates, threadNum * sizeof(hiprandState)));

}

void FreeRender() {
    // free CPU
    free(bvhElem_host);
    free(bound_host);
    free(triangle_host);

    // free GPU
    hipFree(ray_dev);
    hipFree(bvhElem_dev);
    hipFree(material_dev);
    hipFree(triangle_dev);
    hipFree(bound_dev);
    hipFree(frameBuffer);
    hipFree(devStates);
}


__device__ float cudaRandomFloat(hiprandState *state, int pid) {

    hiprandState localState = state[pid];
    hiprand_init((unsigned int) clock64(), pid, 0, &localState);
    return hiprand_uniform(&localState);
}

__device__ void sampleLight(Intersection &pos, float &pdf,
                            int triangleNum, int pid,
                            Triangle* triangles, hiprandState *state) {
    float emit_area_sum = 0;

    // assume we only have one light in the scene
    for (int k = 0; k < triangleNum; k++) {
        if (triangles[k].cudaHasEmit()){
            emit_area_sum += triangles[k].getArea();
        }
    }
    float p = cudaRandomFloat(state, pid) * emit_area_sum;
    emit_area_sum = 0;
    for (int k = 0; k < triangleNum; k++) {
        if (triangles[k].cudaHasEmit()){
            emit_area_sum += triangles[k].getArea();
            if (p <= emit_area_sum){
                triangles[k].CudaSample(pos, pdf, state, pid);
                break;
            }
        }
    }
}

__device__ Intersection SceneIntersect(int pid, Ray ray, int bvhElemNum, int triangleNum,
                                       BVHElem* bvhElems, Triangle* triangles, Bounds3* bounds) {
    Intersection inter;
    inter.coords = Vector3f(-1);
    if (bvhElems == NULL)
        return inter;

    bool visited[32];
    for (int i = 0; i<bvhElemNum; i++) {
        bvhElems[i].visited = false;
        visited[i] = false;
    }


    int arr[3] = {(ray.direction.x <= 0), (ray.direction.y <= 0), (ray.direction.z <= 0)};

    // DFS BVHElem
    int istack[32];
    istack[0] = bvhElems[0].boundIdx;
    int curSize = 1;

    while (curSize > 0) {

        BVHElem &curElem = bvhElems[istack[curSize - 1]];

        bool vl = (curElem.leftIdx < 0 || visited[curElem.leftIdx]);
        bool vr = (curElem.rightIdx < 0 || visited[curElem.rightIdx]);


        if (vl && vr) {
            visited[curElem.boundIdx] = true;
            //curElem.visited = true;
            curSize--;

            if (curElem.isLeaf) { // node is leaf
                for (int a = 0; a < curElem.triNum; a++) { // find intersection with all triangles of this object
                    Intersection ci = triangles[curElem.triStartIdx + a].getIntersection(ray);
                    if (ci.happened && (ci.distance<inter.distance)) {
                        inter = ci;
                    }
                }
            }
        }
        else {
            if (curElem.leftIdx >= 0 && !visited[curElem.leftIdx]) {
//          if (curElem.leftIdx >= 0 && !bvhElems[curElem.leftIdx].visited) {
                BVHElem &left = bvhElems[curElem.leftIdx];
                if (bounds[left.boundIdx].IntersectP(ray,
                                                     Vector3f(1 / (float) ray.direction.x, 1 / (float) ray.direction.y,
                                                              1 / (float) ray.direction.z), arr)) {
                    // if hit left bounding box
                    istack[curSize] = curElem.leftIdx;
                    curSize++;
                } else {
                    visited[left.boundIdx] = true;
                    //left.visited = true;
                }
            }

            if (curElem.rightIdx >= 0 && !visited[curElem.rightIdx]) {
                BVHElem &right = bvhElems[curElem.rightIdx];
                if (bounds[right.boundIdx].IntersectP(ray,
                                                      Vector3f(1 / (float) ray.direction.x, 1 / (float) ray.direction.y,
                                                               1 / (float) ray.direction.z), arr)) {
                    istack[curSize] = curElem.rightIdx;
                    curSize++;
                } else {
                    visited[right.boundIdx] = true;
                    //right.visited = true;
                }
            }

            if (curSize > 32) {
                // TODO: handle CUDA kernel error
                printf("stack overflow %d\n", 32);
            }
        }
    }

    return inter;
}

__device__ Vector3f CalcColor(int pid, int bvhElemNum, int triangleNum,
                              Ray* rays, BVHElem* bvhElems, Triangle* triangles,  Bounds3* bounds, Material* materials,
                              hiprandState *state) {


    Vector3f backgroundColor = Vector3f(0.235294, 0.67451, 0.843137);

    Ray curRay = rays[pid];
    Vector3f pixelColor = Vector3f(0,0,0);
    int maxDepth = 3;  // maxDepth cannot exceed stackSize/2
    float RussianRoulette = 0.8;

    Vector3f vstack[32];
    for (int i = 0; i<32; i++) {
        vstack[i] = Vector3f(0, 0, 0);
    }


    // 这里用stack实现递归，stack内部：
    // 0，1： depth = 0时， in_dir 的颜色 + dir color 需要的系数
    // 2，3： depth = 1时， in_dir 的颜色 + dir color 需要的系数， 以此类推
    int curDepth = 0;
    for (int d = 0; d <= maxDepth; d++) {

        curDepth = d;
        Intersection intersection = SceneIntersect(pid, curRay, bvhElemNum, triangleNum, bvhElems, triangles, bounds);
        if(!intersection.happened) {
            vstack[d*2+0] = backgroundColor;
            break;
        }

        if (intersection.m != NULL && intersection.m->cudaHasEmission()) {
            vstack[d*2+0] = Vector3f(1.0,1.0,1.0);
            break;
        }

        // contribution from the light source
        Vector3f dir_color = Vector3f(0, 0, 0);
        float pdf_light;
        Intersection lightPoint;
        sampleLight(lightPoint, pdf_light, triangleNum, pid, triangles, state);
        lightPoint.normal.normalized();

        Vector3f w_dir = normalize(lightPoint.coords - intersection.coords);
        Ray shadowRay(intersection.coords, w_dir);
        Intersection shadowRayInter = SceneIntersect(pid, shadowRay, bvhElemNum, triangleNum, bvhElems, triangles, bounds);

        // if light ray not blocked in the middle
        if (!shadowRayInter.happened || shadowRayInter.m->cudaHasEmission())
        {
            if (pdf_light < FLT_EPSILON)
                pdf_light = FLT_EPSILON;

            Vector3f f_r1 = intersection.m->eval(-curRay.direction, w_dir, intersection.normal);
            float kk = dotProduct(intersection.coords - lightPoint.coords, intersection.coords - lightPoint.coords);
            dir_color = lightPoint.emit * f_r1 * dotProduct(w_dir, intersection.normal)
                        * dotProduct(-w_dir, lightPoint.normal) / kk / pdf_light;
        }
        vstack[d*2+0] = dir_color;

        // contribution from other objects
        // Russian Roulette
        bool needBreak = true;
        Vector3f indir_color = Vector3f(0,0,0);
        float testrr = cudaRandomFloat(state, pid);
        Vector3f randomDir;
        if (testrr <= RussianRoulette) {
            randomDir = intersection.m->cudaSample(-curRay.direction, intersection.normal, state, pid);
            randomDir = randomDir.normalized();
            float pdf_object = intersection.m->pdf(-curRay.direction, randomDir,intersection.normal);
            Ray ro(intersection.coords, randomDir);

            Intersection objRayInter = SceneIntersect(pid, ro,  bvhElemNum, triangleNum, bvhElems, triangles, bounds);
            if (objRayInter.happened)
                if (!objRayInter.m->cudaHasEmission()) {
                    if (pdf_object < FLT_EPSILON)
                        pdf_object = FLT_EPSILON;

                    Vector3f f_r2 = intersection.m->eval(-curRay.direction, ro.direction, intersection.normal);
                    indir_color = f_r2 * dotProduct(ro.direction, intersection.normal) / pdf_object / RussianRoulette;
                    curRay = ro;
                    needBreak = false;
                }
        }

        vstack[d * 2 + 1] = indir_color;
        if (needBreak)
            break;
    }

    // 这里反过来推算颜色
    for (int i = curDepth; i>0; i--) {
        // in_dir + dir
        Vector3f prev = vstack[i*2] + vstack[i*2+1];
        vstack[2*i-1] = vstack[2*i-1] * prev;
    }
    pixelColor = vstack[0] + vstack[1];


    return pixelColor;
}

__global__ void SetKernelRand(hiprandState *state, int h, int w)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int j = (blockIdx.y * blockDim.y) + threadIdx.y;
    int id = i + j * w;
    /* Each thread gets same seed, a different sequence
       number, no offset */
    hiprand_init(100*sizeof(hiprandState), id, 0, &state[id]);
}

__device__ float deg2rad(const float& deg) { return deg * M_PI / 180.0; }

__global__ void GenerateRay(int width, int height, double fov, float* fb, Ray* rays) {
    Vector3f eye_pos(278, 273, -800);

    float scale = tan(deg2rad(fov * 0.5));
    float imageAspectRatio = width / (float)height;

    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int j = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (i < (float)width && j < (float)height)
    {
        int pixelIdx = i + (j * width);
        float x = (2 * (i + 0.5) / (float)width - 1) *
                  imageAspectRatio * scale;
        float y = (1 - 2 * (j + 0.5) / (float)height) * scale;
        Vector3f dir = normalize(Vector3f(-x, y, 1));

        Ray &ray = rays[pixelIdx];
        ray.origin = eye_pos;
        ray.direction = dir;

    }
}

__global__ void CastRay(int width, int height, float* fb, Ray* rays,
                        int bvhElemNum, int triangleNum,
                        BVHElem* bvhElems, Triangle* triangles, Bounds3* bounds, Material* materials,
                        hiprandState *state) {
    int spp = 8;

    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int j = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (i < width && j < height) {
        int pixelIdx = i + (j * width);
        fb[pixelIdx*3+0] = 0;
        fb[pixelIdx*3+1] = 0;
        fb[pixelIdx*3+2] = 0;
        for (int time = 0; time < spp; time++) {
            Vector3f c = CalcColor(pixelIdx, bvhElemNum, triangleNum,
                                   rays, bvhElems, triangles, bounds, materials, state);
            fb[pixelIdx*3+0] += c.x;
            fb[pixelIdx*3+1] += c.y;
            fb[pixelIdx*3+2] += c.z;
        }

        fb[pixelIdx*3+0] /= (float) spp;
        fb[pixelIdx*3+1] /= (float) spp;
        fb[pixelIdx*3+2] /= (float) spp;


    }
}


__global__ void SetTriangleValue(int triangleNum, Triangle* triangles, Material* materials) {
    for (int i = 0; i<triangleNum; i++) {
        triangles[i].m = materials+i;
    }
}

void Render() {

    SetKernelRand<<<blocks, threads>>>(devStates, scene_host->height, scene_host->width);
    SetTriangleValue<<<blocks, threads>>>(num_triangles, triangle_dev, material_dev);

    GenerateRay<<<blocks, threads>>>(scene_host->width, scene_host->height, scene_host->fov, frameBuffer, ray_dev);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    CastRay<<<blocks, threads>>>(scene_host->width, scene_host->height, frameBuffer,
                                 ray_dev,num_bvhElems, num_triangles,
                                 bvhElem_dev, triangle_dev, bound_dev, material_dev,
                                 devStates);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());


    // save color data to ppm file
    FILE* fp = fopen("image.ppm", "wb");
    (void)fprintf(fp, "P6\n%d %d\n255\n", scene_host->width, scene_host->height);
    for (auto i = 0; i < num_pixles; ++i) {
        static unsigned char color[3];
        color[0] = (unsigned char)(255 * std::pow(clamp(0, 1, frameBuffer[i*3+0]), 0.6f));
        color[1] = (unsigned char)(255 * std::pow(clamp(0, 1, frameBuffer[i*3+1]), 0.6f));
        color[2] = (unsigned char)(255 * std::pow(clamp(0, 1, frameBuffer[i*3+2]), 0.6f));
        fwrite(color, 1, 3, fp);
    }
    fclose(fp);

}

void BuildBvhNodeList(Scene* scene) {

    bvhElem_host = (BVHElem*) malloc(num_bvhElems * sizeof(BVHElem));
    bound_host = (Bounds3*) malloc(num_bvhElems * sizeof(Bounds3));
    std::vector<int> leafIdx;


    for (int i = 0; i<scene->objects.size(); i++) {
        MeshTriangle* mt = (MeshTriangle*)(scene->objects[i]);
        num_triangles += mt->triangles.size();
    }

    triangle_host = (Triangle*) malloc(num_triangles * sizeof(Triangle));

    // BFS for BVH Tree
    BVHBuildNode* root = scene->bvh->root;
    std::queue<BVHBuildNode*> nodeQueue;

    if (root != NULL)
        nodeQueue.push(root);
    int triCount = 0;
    while (!nodeQueue.empty())
    {
        BVHBuildNode* nd = nodeQueue.front();

        BVHElem &curElem = bvhElem_host[nd->nodeIdx];
        curElem.boundIdx = nd->nodeIdx;
        curElem.leftIdx = (nd->left) ? (nd->left->nodeIdx) : -1;
        curElem.rightIdx = (nd->right) ? (nd->right->nodeIdx) : -1;
        curElem.isLeaf = (nd->object);
        if (curElem.isLeaf) {
            MeshTriangle* mt = (MeshTriangle*)(nd->object);
            curElem.triStartIdx = triCount;
            curElem.triNum = mt->triangles.size();
            for (int j = 0; j < curElem.triNum; j++) {
                triangle_host[curElem.triStartIdx + j] = mt->triangles[j];
                triangle_host[curElem.triStartIdx + j].m = mt->m;
                triCount++;
            }
            leafIdx.push_back(curElem.boundIdx);
        }
        else {
            curElem.triStartIdx = -1;
            curElem.triNum = 0;
        }

        bound_host[nd->nodeIdx] = nd->bounds;


        nodeQueue.pop();

        if (nd->left)
        {
            nodeQueue.push(nd->left);
        }
        if (nd->right)
        {
            nodeQueue.push(nd->right);
        }
    }
}



